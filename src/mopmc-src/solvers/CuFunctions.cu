
#include <hip/hip_runtime.h>
//
// Created by guoxin on 8/11/23.
//



//#include <__clang_cuda_builtin_vars.h>
#define cudaAssert(condition) \
    {if (!(condition)){ printf("Assertion %s failed!\n", #condition); asm("trap;"); } \

namespace mopmc{
    namespace functions{
        namespace cuda {
    //namespace functions {
    //    namespace cuda {

            __global__ void aggregate(const double *w, const double *x, double *y, int numRows, int numObjs) {
                // y = x * w
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < numRows) {
                    y[tid] = 0;
                    for (int i = 0; i < numObjs; ++i) {
                        y[tid] += w[i] * x[i * numRows + tid];
                    }
                }
            }

            int aggregateLauncher(const double *w, const double *x, double *y, int numRows, int numObjs) {
                int blockSize, minGridSize, gridSize;
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &aggregate, 0, numRows);
                gridSize = (numRows + blockSize - 1) / blockSize;
                aggregate<<<gridSize, blockSize>>>(w, x, y, numRows, numObjs);
                return 0;
            }

            __global__ void maxValue1(const double *y, double *x, const int *enabledActions,
                                      int *pi, int arrCount, int numRows) {
                // arrCount is the number of states in the model
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < arrCount) {
                    // do some stuff
                    int actionStart = enabledActions[tid];
                    int actionEnd = enabledActions[tid + 1];
                    /*
                    if(tid < arrCount - 1 ) {
                        actionEnd = enabledActions[tid + 1];
                    } else {
                        actionEnd = numRows;
                    }
                     */
                    int maxIndex = pi[tid];
                    double maxValue = y[actionStart + maxIndex];
                    //double maxValue1 = x[tid];
                    for (int action = 0; action < (actionEnd - actionStart); ++action) {
                        if (y[actionStart + action] > maxValue) {
                            maxIndex = action;
                            maxValue = y[actionStart + action];
                        }
                    }
                    x[tid] = maxValue;
                    pi[tid] = maxIndex;
                }
            }

            int maxValueLauncher1(double *y, double *x, int *enabledActions, int *pi, int arrCount, int numRows) {
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &maxValue1, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;

                maxValue1<<<gridSize, blockSize>>>(y, x, enabledActions, pi, arrCount, numRows);
                return 0;
            }


            __global__ void row2RowGroup(const int *row2RowGroupMapping, int *x, int arrCount) {
                //arrCount == B_nnz
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < arrCount) {
                    int rowInd = x[tid];
                    int rowGroupInd = row2RowGroupMapping[rowInd];
                    x[tid] = rowGroupInd;
                }
            }

            int row2RowGroupLauncher(const int *row2RowGroupMapping, int *x, int arrCount) {
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &row2RowGroup, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;
                row2RowGroup<<<gridSize, blockSize>>>(row2RowGroupMapping, x, arrCount);
                return 0;
            }


            __global__ void binaryMasking(const int *csrOffsets, const int *rowGroupIndices, const int *row2RowGroupMapping,
                                          const int *pi, int *masking4rows, int *masking4nzz, int arrCount) {
                //arrCount == nrows
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < arrCount) {
                    int rowGroupInd = row2RowGroupMapping[tid];
                    int firstRowInRowGroup = rowGroupIndices[rowGroupInd];
                    int selectedActionInd = pi[rowGroupInd];
                    int val = 0;
                    if (tid == firstRowInRowGroup + selectedActionInd) {
                        val = 1;
                    }
                    masking4rows[tid] = val;
                    int start = csrOffsets[tid];
                    int incr = csrOffsets[tid + 1] - start;
                    for (int i = 0; i < incr; ++i) {
                        masking4nzz[start + i] = val;
                    }

                }
            }


            int binaryMaskingLauncher(const int* csrOffsets, const int *rowGroupIndices, const int *row2RowGroupMapping,
                                      const int* pi, int *masking4rows, int* masking4nnz, int arrCount) {
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &binaryMasking, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;
                binaryMasking<<<gridSize, blockSize>>>(csrOffsets, rowGroupIndices, row2RowGroupMapping,
                                                       pi, masking4rows, masking4nnz, arrCount);
                return 0;
            }

            __global__ void tiling(const int* mask, int* tiledMask, const int ncopies, const int n) {
                // basically just copy the mask ncopies times. 
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < n) {
                    for (int i = 0; i < ncopies; ++i) {
                        tiledMask[i * n + tid] = mask[tid];
                    }
                }
            }
            
            int tilingLauncher(const int* mask, int* tiledMask, const int ncopies, const int n) {
                int blockSize, minGridSize, gridSize;                

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, tiling, 0, n);

                gridSize = (n + blockSize - 1) / blockSize;
                tiling<<<gridSize, blockSize>>>(mask, tiledMask, ncopies, n);
                return 0;
            }

            __global__ void maxValue2(const double *y, double *x, const int *enabledActions,
                                      int *pi, int *bpi, int arrCount) {
                // arrCount is the number of states in the model
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < arrCount) {
                    // do some stuff
                    int actionStart = enabledActions[tid];
                    int actionEnd = enabledActions[tid + 1];
                    int maxIndex = pi[tid];
                    double maxValue = y[actionStart + maxIndex];
                    //update pi and x
                    for (int action = 0; action < (actionEnd - actionStart); ++action) {
                        if (y[actionStart + action] > maxValue) {
                            maxIndex = action;
                            maxValue = y[actionStart + action];
                        }
                    }
                    x[tid] = maxValue;
                    pi[tid] = maxIndex;

                    //update binary pi
                    for (int action = 0; action < (actionEnd - actionStart); ++action) {
                        if (action == maxIndex) {
                            bpi[actionStart + action] = 1;
                        } else {
                            bpi[actionStart + action] = 0;
                        }
                    }
                }
            }

            int maxValueLauncher2(double *y, double *x, int *enabledActions, int *pi, int *bpi, int arrCount) {
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &maxValue2, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;

                maxValue2<<<gridSize, blockSize>>>(y, x, enabledActions, pi, bpi, arrCount);
                return 0;
            }


            __global__ void abs(const double *x, int k) {
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                double diff = 0.0;
                if (tid < k) {
                    if (diff < x[tid]) {
                        diff = x[tid];
                    }
                    if (diff < -1.0 * x[tid])
                        diff = x[tid];
                }
            }

            int absLauncher(const double *x, int k) {
                int blockSize, minGridSize, gridSize;
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &abs, 0, k);
                gridSize = (blockSize - 1) / blockSize;
                abs<<<gridSize, blockSize>>>(x, k);
                return 0;
            }

        }
    }
}